#include <hip/hip_runtime.h>
#include <math.h>
#include <chrono>
#include <cstdlib>
#include <ctime>
#include <iostream>

__global__ void sum_vectors(double* a, double* b, double* c, int n) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    c[idx] = a[idx] + b[idx];
  }
}

bool check(double* a, double* b, double* c, int n) {
  for (int i = 0; i < n; i++) {
    if (c[i] != a[i] + b[i]) {
      return false;
    }
  }
  return true;
}

int main(int argc, char** argv) {
  // Size of vectors
  int n = atoi(argv[1]);
  // Host vectors
  double *h_a, *h_b, *h_c;
  // Size, in bytes, of each vector
  size_t bytes = n * sizeof(double);
  // Allocate memory for each vector on host
  h_a = (double*)malloc(bytes);
  h_b = (double*)malloc(bytes);
  h_c = (double*)malloc(bytes);
  // Initialize vectors on host
  for (int i = 0; i < n; i++) {
    h_a[i] = sin(i) * sin(i);
    h_b[i] = cos(i) * cos(i);
  }
  // Device input vectors
  double *d_a, *d_b, *d_c;
  // Size, in bytes, of each vector
  // size_t bytes = n * sizeof(double);
  // Allocate memory for each vector on GPU
  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);
  // Copy host vectors to device
  hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

  // -------------------------------------

  // Number of threads in each thread block
  int blockSize = 1024;
  // Number of thread blocks in grid
  int gridSize = (n - 1) / blockSize + 1;
  // create gpu timer
  hipEvent_t start_gpu, stop_gpu;
  hipEventCreate(&start_gpu);
  hipEventCreate(&stop_gpu);
  hipEventRecord(start_gpu);
  // Execute the kernel
  sum_vectors<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
  // get gpu time
  hipDeviceSynchronize();
  hipEventRecord(stop_gpu);
  float time_gpu = 0;
  hipEventElapsedTime(&time_gpu, start_gpu, stop_gpu);
  // Copy array back to host
  hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

  /*// create cpu timer
  auto start_cpu = high_resolution_clock::now();
  // check result
  bool res = check(h_a, h_b, h_c, n);
  // get cpu time
  auto stop_cpu = high_resolution_clock::now();
  auto time_cpu = stop_cpu - start_cpu;
  time_cpu = time_cpu.count() / 1000;*/

  // create cpu timer
  hipEvent_t start_cpu, stop_cpu;
  hipEventCreate(&start_cpu);
  hipEventCreate(&stop_cpu);
  hipEventRecord(start_cpu);
  // check result
  bool res = check(h_a, h_b, h_c, n);
  // get cpu time
  hipDeviceSynchronize();
  hipEventRecord(stop_cpu);
  float time_cpu = 0;
  hipEventElapsedTime(&time_cpu, start_cpu, stop_cpu);


  if (res) {
    printf("correct\n");
  } else {
    printf("incorrect\n");
  }

  printf("CPU time: %10f ms\n", time_cpu);
  printf("GPU time: %10f ms\n", time_gpu);

  // Release device memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  // Release host memory
  free(h_a);
  free(h_b);
  free(h_c);
  return 0;
}

/*cudaEvent_t start, stop;
cudaEventCreate(&start);
cudaEventCreate(&stop);
cudaMemcpy(d_x, x, N * sizeof(float), cudaMemcpyHostToDevice);
cudaMemcpy(d_y, y, N * sizeof(float), cudaMemcpyHostToDevice);
cudaEventRecord(start);
saxpy<<<(N + 255) / 256, 256>>>(N, 2.0f, d_x, d_y);
cudaDeviceSynchronize();
cudaEventRecord(stop);
cudaMemcpy(y, d_y, N * sizeof(float), cudaMemcpyDeviceToHost);
cudaEventSynchronize(stop);
float milliseconds = 0;
cudaEventElapsedTime(&milliseconds, start, stop);*/
