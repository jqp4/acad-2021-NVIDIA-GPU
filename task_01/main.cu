#include <hip/hip_runtime.h>
#include <math.h>
#include <chrono>
#include <cstdlib>
#include <ctime>
#include <iostream>

__global__ void sum_vectors(double* a, double* b, double* c, int n) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    c[idx] = a[idx] + b[idx];
  }
}

bool check(double* a, double* b, double* c, int n) {
  for (int i = 0; i < n; i++) {
    if (c[i] != a[i] + b[i]) {
      return false;
    }
  }
  return true;
}

int main(int argc, char** argv) {
  // Size of vectors
  int n = atoi(argv[1]);
  // Host vectors
  double *h_a, *h_b, *h_c;
  // Size, in bytes, of each vector
  size_t bytes = n * sizeof(double);
  // Allocate memory for each vector on host
  h_a = (double*)malloc(bytes);
  h_b = (double*)malloc(bytes);
  h_c = (double*)malloc(bytes);
  // Initialize vectors on host
  for (int i = 0; i < n; i++) {
    h_a[i] = sin(i) * sin(i);
    h_b[i] = cos(i) * cos(i);
  }
  // Device input vectors
  double *d_a, *d_b, *d_c;
  // Size, in bytes, of each vector
  // size_t bytes = n * sizeof(double);
  // Allocate memory for each vector on GPU
  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);
  // Copy host vectors to device
  hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

  // -------------------------------------

  // Number of threads in each thread block
  int blockSize = 1024;
  // Number of thread blocks in grid
  int gridSize = (n - 1) / blockSize + 1;
  // Execute the kernel
  sum_vectors<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
  // --------------------------
  hipDeviceSynchronize();
  // Copy array back to host
  hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

  bool res = check(h_a, h_b, h_c, n);

  if (res) {
    printf("correct\n");
  } else {
    printf("incorrect\n");
  }
  // Release device memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  free(h_a);
  free(h_b);
  free(h_c);
  return 0;
}

/*cudaEvent_t start, stop;
cudaEventCreate(&start);
cudaEventCreate(&stop);
cudaMemcpy(d_x, x, N * sizeof(float), cudaMemcpyHostToDevice);
cudaMemcpy(d_y, y, N * sizeof(float), cudaMemcpyHostToDevice);
cudaEventRecord(start);
saxpy<<<(N + 255) / 256, 256>>>(N, 2.0f, d_x, d_y);
cudaDeviceSynchronize();
cudaEventRecord(stop);
cudaMemcpy(y, d_y, N * sizeof(float), cudaMemcpyDeviceToHost);
cudaEventSynchronize(stop);
float milliseconds = 0;
cudaEventElapsedTime(&milliseconds, start, stop);*/
